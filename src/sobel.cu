#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void sobel_kernel(const unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && y >= 1 && x < width - 1 && y < height - 1) {
        int gx = -input[(y - 1) * width + (x - 1)] - 2 * input[y * width + (x - 1)] - input[(y + 1) * width + (x - 1)]
                 + input[(y - 1) * width + (x + 1)] + 2 * input[y * width + (x + 1)] + input[(y + 1) * width + (x + 1)];

        int gy = -input[(y - 1) * width + (x - 1)] - 2 * input[(y - 1) * width + x] - input[(y - 1) * width + (x + 1)]
                 + input[(y + 1) * width + (x - 1)] + 2 * input[(y + 1) * width + x] + input[(y + 1) * width + (x + 1)];

        int magnitude = min(255, (int)sqrtf((float)(gx * gx + gy * gy)));
        output[y * width + x] = (unsigned char)magnitude;
    }
}


bool readPGM(const std::string& filename, std::vector<unsigned char>& data, int& width, int& height) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) return false;

    std::string magic;
    file >> magic;
    if (magic != "P5") return false;

    file >> width >> height;
    int maxval;
    file >> maxval;
    file.ignore(1);  // skip newline

    data.resize(width * height);
    file.read(reinterpret_cast<char*>(data.data()), data.size());

    return true;
}

bool writePGM(const std::string& filename, const std::vector<unsigned char>& data, int width, int height) {
    std::ofstream file(filename, std::ios::binary);
    if (!file) return false;

    file << "P5\n" << width << " " << height << "\n255\n";
    file.write(reinterpret_cast<const char*>(data.data()), data.size());

    return true;
}


int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: ./sobel input.pgm output.pgm\n";
        return 1;
    }

    std::string inputPath = argv[1], outputPath = argv[2];
    std::vector<unsigned char> inputImage, outputImage;
    int width, height;

    if (!readPGM(inputPath, inputImage, width, height)) {
        std::cerr << "Error reading input image\n";
        return 1;
    }

    outputImage.resize(width * height);

    unsigned char *d_input, *d_output;
    size_t size = width * height * sizeof(unsigned char);

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);
    hipMemcpy(d_input, inputImage.data(), size, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    sobel_kernel<<<gridDim, blockDim>>>(d_input, d_output, width, height);
    hipDeviceSynchronize();

    hipMemcpy(outputImage.data(), d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    if (!writePGM(outputPath, outputImage, width, height)) {
        std::cerr << "Error writing output image\n";
        return 1;
    }

    std::cout << "Edge detection completed successfully.\n";
    return 0;
}